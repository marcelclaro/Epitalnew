#include "hip/hip_runtime.h"
/*
 * CUDAfunctions.cu
 *
 *  Created on: Nov 27, 2014
 *      Author: marcel
 */

#include "CUDAfunctions.hpp"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <cmath>



__device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__double_as_longlong(val + __longlong_as_double(assumed)));
		// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
		} while (assumed != old);

	return __longlong_as_double(old);
}

__global__ void modulus(hipDoubleComplex* initialpptr, long int dimx,long int dimy,long int dimz, double* value){
	long int xmax=dimx-1;
	long int ymax=dimy-1;
	long int zmax=dimz-1;
	double partial=0.0;

	long int l = threadIdx.x;
	long int m = blockIdx.x;

        char* pointer = (char*)initialpptr; 
	size_t pitch = dimx*sizeof(hipDoubleComplex);
	size_t slicePitch = pitch * dimy;
	char* slice = pointer + m * slicePitch;
	hipDoubleComplex* initial=(hipDoubleComplex*)(slice + l * pitch);


	if(l==0 || l ==ymax){
		partial+=hipCreal(hipCmul(initial[0],hipConj(initial[0])))/4.0;
		for(long int k = 1; k < xmax; ++k){
			partial+=hipCreal(hipCmul(initial[k],hipConj(initial[k])))/2.0;
		}
		partial+=hipCreal(hipCmul(initial[xmax],hipConj(initial[xmax])))/4.0;
		if(m==0 || m ==zmax){
			partial+=hipCreal(hipCmul(initial[0],hipConj(initial[0])))/8.0;
			for(long int k = 1; k < xmax; ++k){
				partial+=hipCreal(hipCmul(initial[k],hipConj(initial[k])))/4.0;
			}
			partial+=hipCreal(hipCmul(initial[xmax],hipConj(initial[xmax])))/8.0;
		}
	}
	else if(m==0 || m ==zmax){
		partial+=hipCreal(hipCmul(initial[0],hipConj(initial[0])))/4.0;
		for(long int k = 1; k < xmax; ++k){
			partial+=hipCreal(hipCmul(initial[k],hipConj(initial[k])))/2.0;
		}
		partial+=hipCreal(hipCmul(initial[xmax],hipConj(initial[xmax])))/4.0;
	}

	partial+=hipCreal(hipCmul(initial[0],hipConj(initial[0])))/2.0;
	for(long int k = 1; k < xmax; ++k){
		partial+=hipCreal(hipCmul(initial[k],hipConj(initial[k])));
	}
	partial+=hipCreal(hipCmul(initial[xmax],hipConj(initial[xmax])))/2.0;


	atomicAdd(value,partial);
}


__global__ void dividebyscalar3D(hipDoubleComplex* initialpptr, long int dimx,long int dimy,long int dimz, double value){
	long int xmax=dimx-1;

	long int l = threadIdx.x;
	long int m = blockIdx.x;

	char* pointer = (char*)initialpptr;
	size_t pitch = dimx*sizeof(hipDoubleComplex);
	size_t slicePitch = pitch * dimy;
	char* slice = pointer + m * slicePitch;
	hipDoubleComplex* initial=(hipDoubleComplex*)(slice + l * pitch);
	hipDoubleComplex complexvalue = make_hipDoubleComplex(value,0.0);

	for(long int k = 0; k <= xmax; ++k){
		initial[k]=hipCdiv(initial[k],complexvalue);
	}

}

__global__ void matrixmulti3D(hipDoubleComplex* initialpptr, hipDoubleComplex* potentialpptr, long int dimx,long int dimy,long int dimz){
	long int xmax=dimx-1;


	long int l = threadIdx.x;
	long int m = blockIdx.x;

	char* pointer = (char*)initialpptr;
	size_t pitch = dimx*sizeof(hipDoubleComplex);
	size_t slicePitch = pitch * dimy;
	char* slice = pointer + m * slicePitch;
	hipDoubleComplex* initial=(hipDoubleComplex*)(slice + l * pitch);

	char* pointertwo = (char*)potentialpptr;
	size_t pitchtwo = dimx*sizeof(hipDoubleComplex);
	size_t slicePitchtwo = pitchtwo * dimy;
	char* slicetwo = pointertwo + m * slicePitchtwo;
	hipDoubleComplex* potential=(hipDoubleComplex*)(slicetwo + l * pitchtwo);

	for(long int k = 0; k <= xmax; ++k){
		initial[k]=hipCmul(initial[k],potential[k]);
	}

}

__host__ void getmodulus3D(hipDoubleComplex* initialpptr, long int dimx,long int dimy,long int dimz, double xstep, double ystep, double zstep, double* modulusout){
	double* modulusinside;
        hipMalloc(&modulusinside,sizeof(double));
	double zero = 0.0;
	hipMemcpy(modulusinside, &zero, sizeof(double),hipMemcpyHostToDevice);
	modulus<<<dimz,dimy>>>(initialpptr,dimx,dimy,dimz,modulusinside);
	double value;
	hipMemcpy(&value, modulusinside, sizeof(double),hipMemcpyDeviceToHost);
        value=sqrt(value*(dimx-1)*(dimy-1)*(dimy-1)*xstep*ystep*zstep);
        *modulusout=value;
        hipFree(modulusinside);
}

 void groundlevel_periodicFFT_CUDA(long int timesteps,std::complex<double>* initial,std::complex<double>* potential,std::complex<double>* fftmulti,
		double effectivemass, long int dimx,long int dimy,long int dimz, double xstep, double ystep, double zstep){
	double modulusvalue;

	size_t sizeComplex = dimx*dimy*dimz*sizeof(hipDoubleComplex);

	hipDoubleComplex* initialPtr;
	hipDoubleComplex* potentialPtr;
	hipDoubleComplex* fftmultiPtr;

	hipMalloc(&initialPtr, sizeComplex);
	if (hipGetLastError() != hipSuccess){
	 fprintf(stderr, "Cuda error: Failed to allocate\n");
	return;
	}

	hipMalloc(&potentialPtr, sizeComplex);
	if (hipGetLastError() != hipSuccess){
	 fprintf(stderr, "Cuda error: Failed to allocate\n");
	return;
	}

	hipMalloc(&fftmultiPtr, sizeComplex);
	if (hipGetLastError() != hipSuccess){
	 fprintf(stderr, "Cuda error: Failed to allocate\n");
	return;
	}

        hipMemcpy(initialPtr,initial,sizeComplex,hipMemcpyHostToDevice);
	if (hipGetLastError() != hipSuccess){
	 fprintf(stderr, "Cuda error: Failed to copy to device\n");
	return;
	}	
        hipMemcpy(potentialPtr,potential,sizeComplex,hipMemcpyHostToDevice);
        	if (hipGetLastError() != hipSuccess){
	 fprintf(stderr, "Cuda error: Failed to copy to device\n");
	return;
	}
        hipMemcpy(fftmultiPtr,fftmulti,sizeComplex,hipMemcpyHostToDevice);
        	if (hipGetLastError() != hipSuccess){
	 fprintf(stderr, "Cuda error: Failed to copy to device\n");
	return;
	}


        
	getmodulus3D(initialPtr,dimx,dimy,dimz,xstep,ystep,zstep,&modulusvalue);  
	dividebyscalar3D<<<dimz,dimy>>>(initialPtr,dimx,dimy,dimz,modulusvalue);
                
	hipfftHandle plan;
	int n[3] = {dimx,dimy,dimz};

        //int inembed[] = {dimx,dimy,initialPitchedPtr.pitch/sizeof(hipfftComplex)}; // Input size with pitch
        //int onembed[] = {dimx,dimy,initialPitchedPtr.pitch/sizeof(hipfftComplex)}; // Output size with pitch

	/* Create a 3D FFT plan. */
	if (hipfftPlanMany(&plan, 3, n,
	 NULL, 1, 1, // *inembed, istride, idist
	 NULL, 1, 1, // *onembed, ostride, odist
	 HIPFFT_Z2Z, 1) != HIPFFT_SUCCESS){
	 fprintf(stderr, "CUFFT error: Plan creation failed");
	return;
	}

     
        
        
	for(long int i = 0 ; i< timesteps; i++){
		printf("step: %i...\n",i);
		matrixmulti3D<<<dimz,dimy>>>(initialPtr,potentialPtr,dimx,dimy,dimz);
		hipfftExecZ2Z(plan,(hipfftDoubleComplex*)initialPtr, (hipfftDoubleComplex*)initialPtr, HIPFFT_FORWARD);
		matrixmulti3D<<<dimz,dimy>>>(initialPtr,fftmultiPtr,dimx,dimy,dimz);
		hipfftExecZ2Z(plan,(hipfftDoubleComplex*)initialPtr , (hipfftDoubleComplex*)initialPtr, HIPFFT_BACKWARD);
		matrixmulti3D<<<dimz,dimy>>>(initialPtr,potentialPtr,dimx,dimy,dimz);
		double tonorm;
                getmodulus3D(initialPtr,dimx,dimy,dimz,xstep,ystep,zstep,&tonorm);
		dividebyscalar3D<<<dimz,dimy>>>(initialPtr,dimx,dimy,dimz,tonorm);
	}
        
                
        hipMemcpy(initial,initialPtr,sizeComplex,hipMemcpyDeviceToHost);
	hipFree(initialPtr);
	hipFree(potentialPtr);
	hipFree(fftmultiPtr);

}


